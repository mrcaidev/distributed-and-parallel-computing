#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"
#include <hip/hip_runtime.h>

#define SOFTENING 1e-9f
#define BLOCK_SIZE 32

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct
{
    float x, y, z, vx, vy, vz;
} Body;

/*
 * Do not modify this function. A constraint of this exercise is
 * that it remain a host function.
 */

void randomizeBodies(float *data, int n)
{
    for (int i = 0; i < n; i++)
    {
        data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    }
}

/*
 * This function calculates the gravitational impact of all bodies in the system
 * on all others, but does not update their positions.
 */

__global__ void bodyForce(Body *p, float dt, int n)
{
    // 计算本线程负责的 body 下标。
    int i = threadIdx.x + blockIdx.x / BLOCK_SIZE * blockDim.x;

    if (i >= n)
    {
        return;
    }

    // 块级共享内存，用于加速该块内所有线程的受力计算。
    __shared__ float3 tile[BLOCK_SIZE];

    float Fx = 0.0f;
    float Fy = 0.0f;
    float Fz = 0.0f;

    for (int k = blockIdx.x % BLOCK_SIZE; k < n / BLOCK_SIZE; k += BLOCK_SIZE)
    {
        // 向共享内存装入新一批 body。
        Body temp = p[k * BLOCK_SIZE + threadIdx.x];
        tile[threadIdx.x] = make_float3(temp.x, temp.y, temp.z);

        // 等待所有线程装入完毕。
        __syncthreads();

        // 叠加计算共享内存内所有 body 施加的力。
        for (int j = 0; j < BLOCK_SIZE; j++)
        {
            float dx = tile[j].x - p[i].x;
            float dy = tile[j].y - p[i].y;
            float dz = tile[j].z - p[i].z;
            float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;

            Fx += dx * invDist3;
            Fy += dy * invDist3;
            Fz += dz * invDist3;
        }

        // 等待所有线程使用完共享内存内的 body。
        __syncthreads();
    }

    // 用原子计算更新速度，避免竞态问题。
    atomicAdd(&p[i].vx, dt * Fx);
    atomicAdd(&p[i].vy, dt * Fy);
    atomicAdd(&p[i].vz, dt * Fz);
}

__global__ void integratePosition(Body *p, float dt, int n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i >= n)
    {
        return;
    }

    p[i].x += p[i].vx * dt;
    p[i].y += p[i].vy * dt;
    p[i].z += p[i].vz * dt;
}

int main(const int argc, const char **argv)
{

    /*
     * Do not change the value for `nBodies` here. If you would like to modify it,
     * pass values into the command line.
     */

    int nBodies = 2 << 11;
    int salt = 0;
    if (argc > 1)
        nBodies = 2 << atoi(argv[1]);

    /*
     * This salt is for assessment reasons. Tampering with it will result in automatic failure.
     */

    if (argc > 2)
        salt = atoi(argv[2]);

    const float dt = 0.01f; // time step
    const int nIters = 10;  // simulation iterations

    int bytes = nBodies * sizeof(Body);
    float *buf;
    hipHostMalloc(&buf, bytes);

    /*
     * As a constraint of this exercise, `randomizeBodies` must remain a host function.
     */

    randomizeBodies(buf, 6 * nBodies); // Init pos / vel data

    float *d_buf;
    hipMalloc(&d_buf, bytes);

    Body *d_p = (Body *)d_buf;
    hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);

    int nBlocks = (nBodies - 1) / BLOCK_SIZE + 1;

    double totalTime = 0.0;

    /*
     * This simulation will run for 10 cycles of time, calculating gravitational
     * interaction amongst bodies, and adjusting their positions to reflect.
     */

    /*******************************************************************/
    // Do not modify these 2 lines of code.
    for (int iter = 0; iter < nIters; iter++)
    {
        StartTimer();
        /*******************************************************************/

        /*
         * You will likely wish to refactor the work being done in `bodyForce`,
         * as well as the work to integrate the positions.
         */

        bodyForce<<<nBodies, BLOCK_SIZE>>>(d_p, dt, nBodies); // compute interbody forces

        /*
         * This position integration cannot occur until this round of `bodyForce` has completed.
         * Also, the next round of `bodyForce` cannot begin until the integration is complete.
         */

        integratePosition<<<nBlocks, BLOCK_SIZE>>>(d_p, dt, nBodies);

        if (iter == nIters - 1)
        {
            hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);
        }

        /*******************************************************************/
        // Do not modify the code in this section.
        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }

    double avgTime = totalTime / (double)(nIters);
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
    checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
    checkAccuracy(buf, nBodies);
    printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
    salt += 1;
#endif
    /*******************************************************************/

    /*
     * Feel free to modify code below.
     */

    hipFree(d_buf);
    hipHostFree(buf);
}
