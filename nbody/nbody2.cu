#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"
#include <hip/hip_runtime.h>

#define SOFTENING 1e-9f
#define BLOCK_SIZE 128
#define BLOCK_STEP 32

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct
{
    float x, y, z, vx, vy, vz;
} Body;

/*
 * Do not modify this function. A constraint of this exercise is
 * that it remain a host function.
 */

void randomizeBodies(float *data, int n)
{
    for (int i = 0; i < n; i++)
    {
        data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    }
}

/*
 * This function calculates the gravitational impact of all bodies in the system
 * on all others, but does not update their positions.
 */

__global__ void bodyForce(Body *p, float dt, int n)
{
    // 从全局内存获取本线程负责的物体。
    int i = (threadIdx.x + blockIdx.x * blockDim.x) % n;
    Body body = p[i];

    // 块级共享内存，用于缓存一个批次的施力物体。
    __shared__ float3 tile[BLOCK_SIZE];

    float Fx = 0.0f;
    float Fy = 0.0f;
    float Fz = 0.0f;

    int nBlocks = n / BLOCK_SIZE;
    int k = blockIdx.x + blockIdx.x / nBlocks;

#pragma unroll 32
    for (int swap = 0; swap < n / (BLOCK_STEP * BLOCK_SIZE); swap++)
    {
        k %= nBlocks;

        // 从全局内存获取新一批物体，装入共享内存。
        Body temp = p[k * BLOCK_SIZE + threadIdx.x];
        tile[threadIdx.x] = make_float3(temp.x, temp.y, temp.z);

        // 确保新一批物体已经全部装入。
        __syncthreads();

#pragma unroll 32
        // 叠加新一批物体施加在负责物体上的引力。
        for (int j = 0; j < BLOCK_SIZE; j++)
        {
            float dx = tile[j].x - body.x;
            float dy = tile[j].y - body.y;
            float dz = tile[j].z - body.z;
            float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;

            Fx += dx * invDist3;
            Fy += dy * invDist3;
            Fz += dz * invDist3;
        }

        // 确保新一批物体已经全部消耗。
        __syncthreads();

        k += BLOCK_STEP;
    }

    // 使用原子加法更新速度，避免竞态问题。
    atomicAdd(&p[i].vx, dt * Fx);
    atomicAdd(&p[i].vy, dt * Fy);
    atomicAdd(&p[i].vz, dt * Fz);
}

__global__ void integratePosition(Body *p, float dt, int n)
{
    // 计算本线程负责的物体的下标。
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i >= n)
    {
        return;
    }

    // 更新坐标。
    p[i].x += p[i].vx * dt;
    p[i].y += p[i].vy * dt;
    p[i].z += p[i].vz * dt;
}

int main(const int argc, const char **argv)
{

    /*
     * Do not change the value for `nBodies` here. If you would like to modify it,
     * pass values into the command line.
     */

    int nBodies = 2 << 11;
    int salt = 0;
    if (argc > 1)
        nBodies = 2 << atoi(argv[1]);

    /*
     * This salt is for assessment reasons. Tampering with it will result in automatic failure.
     */

    if (argc > 2)
        salt = atoi(argv[2]);

    const float dt = 0.01f; // time step
    const int nIters = 10;  // simulation iterations

    int bytes = nBodies * sizeof(Body);
    float *buf;
    hipHostMalloc(&buf, bytes);

    /*
     * As a constraint of this exercise, `randomizeBodies` must remain a host function.
     */

    randomizeBodies(buf, 6 * nBodies); // Init pos / vel data

    float *d_buf;
    hipMalloc(&d_buf, bytes);
    Body *d_p = (Body *)d_buf;
    hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);

    int nBlocks = nBodies / BLOCK_SIZE;

    double totalTime = 0.0;

    /*
     * This simulation will run for 10 cycles of time, calculating gravitational
     * interaction amongst bodies, and adjusting their positions to reflect.
     */

    /*******************************************************************/
    // Do not modify these 2 lines of code.
    for (int iter = 0; iter < nIters; iter++)
    {
        StartTimer();
        /*******************************************************************/

        /*
         * You will likely wish to refactor the work being done in `bodyForce`,
         * as well as the work to integrate the positions.
         */

        bodyForce<<<nBlocks * BLOCK_STEP, BLOCK_SIZE>>>(d_p, dt, nBodies); // compute interbody forces

        /*
         * This position integration cannot occur until this round of `bodyForce` has completed.
         * Also, the next round of `bodyForce` cannot begin until the integration is complete.
         */

        integratePosition<<<nBlocks, BLOCK_SIZE>>>(d_p, dt, nBodies);

        if (iter == nIters - 1)
        {
            hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);
        }

        /*******************************************************************/
        // Do not modify the code in this section.
        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }

    double avgTime = totalTime / (double)(nIters);
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
    checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
    checkAccuracy(buf, nBodies);
    printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
    salt += 1;
#endif
    /*******************************************************************/

    /*
     * Feel free to modify code below.
     */

    hipFree(d_buf);
    hipHostFree(buf);
}
